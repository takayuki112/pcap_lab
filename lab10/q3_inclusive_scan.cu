#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256

__global__ void scan_kernel(int* d_in, int* d_out, int n) {
    extern __shared__ int temp[];
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;
    
    temp[tid] = (gid < n) ? d_in[gid] : 0;
    __syncthreads();
    
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int val = (tid >= stride) ? temp[tid - stride] : 0;
        __syncthreads();
        if (tid >= stride) temp[tid] += val;
        __syncthreads();
    }
    
    if (gid < n) d_out[gid] = temp[tid];
}

int main() {
    int n;
    printf("Enter array size: ");
    scanf("%d", &n);
    
    int* h_in = (int*)malloc(n * sizeof(int));
    int* h_out = (int*)malloc(n * sizeof(int));
    
    printf("Enter %d integers:\n", n);
    for (int i = 0; i < n; i++) scanf("%d", &h_in[i]);
    
    int *d_in, *d_out;
    hipMalloc(&d_in, n * sizeof(int));
    hipMalloc(&d_out, n * sizeof(int));
    
    hipMemcpy(d_in, h_in, n * sizeof(int), hipMemcpyHostToDevice);
    
    dim3 block(THREADS_PER_BLOCK);
    dim3 grid((n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    scan_kernel<<<grid, block, THREADS_PER_BLOCK * sizeof(int)>>>(d_in, d_out, n);
    
    hipMemcpy(h_out, d_out, n * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("\nInclusive scan result:\n");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_out[i]);
    }
    printf("\n");
    
    free(h_in); free(h_out);
    hipFree(d_in); hipFree(d_out);
    
    return 0;
}