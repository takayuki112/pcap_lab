#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define TILE_WIDTH 2

__global__ void matmul_shared(int *a, int *b, int *c, int ha, int wahb, int wb) {
    __shared__ int sharedA[TILE_WIDTH][TILE_WIDTH];
    __shared__ int sharedB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    int sum = 0;

    for (int t = 0; t < (wahb + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
        int a_col = t * TILE_WIDTH + threadIdx.x;
        int b_row = t * TILE_WIDTH + threadIdx.y;
        
        sharedA[threadIdx.y][threadIdx.x] = (row < ha && a_col < wahb) ? a[row * wahb + a_col] : 0;
        sharedB[threadIdx.y][threadIdx.x] = (b_row < wahb && col < wb) ? b[b_row * wb + col] : 0;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) {
            sum += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < ha && col < wb) {
        c[row * wb + col] = sum;
    }
}

int main(void) {
    int ha, wahb, wb, i;
    int *a, *b, *c, *da, *db, *dc;

    printf("Enter 3 dimensions ha, wa=hb, wb: \n");
    scanf("%d%d%d", &ha, &wahb, &wb);

    a = (int*)malloc(sizeof(int) * ha * wahb);
    b = (int*)malloc(sizeof(int) * wahb * wb);
    c = (int*)malloc(sizeof(int) * ha * wb);

    printf("Enter Matrix A (%d elements): \n", ha * wahb);
    for (i = 0; i < ha * wahb; i++) scanf("%d", &a[i]);

    printf("Enter Matrix B (%d elements): \n", wahb * wb);
    for (i = 0; i < wahb * wb; i++) scanf("%d", &b[i]);

    hipMalloc((void**)&da, sizeof(int) * ha * wahb);
    hipMalloc((void**)&db, sizeof(int) * wahb * wb);
    hipMalloc((void**)&dc, sizeof(int) * ha * wb);

    hipMemcpy(da, a, sizeof(int) * ha * wahb, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int) * wahb * wb, hipMemcpyHostToDevice);

    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    dim3 gridSize((wb + blockSize.x - 1) / blockSize.x,
                  (ha + blockSize.y - 1) / blockSize.y);

    matmul_shared<<<gridSize, blockSize>>>(da, db, dc, ha, wahb, wb);

    hipMemcpy(c, dc, sizeof(int) * ha * wb, hipMemcpyDeviceToHost);

    printf("Resultant matrix: \n");
    for (i = 0; i < ha * wb; i++) {
        if (i % wb == 0) printf("\n");
        printf("%d \t", c[i]);
    }
    printf("\n");

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);

    return 0;
}