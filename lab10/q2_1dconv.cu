#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MAX_K 256
#define THREADS_PER_BLOCK 256

__constant__ int dk[MAX_K];

__global__ void conv1d(int* in, int* out, int n, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n) {
        int r = 0;
        int ks = max(0, k - 1 - i);
        int ke = min(k, n - i + k - 1);
        
        for (int j = ks; j < ke; j++) {
            r += in[i - k + 1 + j] * dk[j];
        }
        out[i] = r;
    }
}

int main() {
    int n, k;
    
    printf("Input size: ");
    scanf("%d", &n);
    printf("Kernel size (max %d): ", MAX_K);
    scanf("%d", &k);
    
    if (k > MAX_K) {
        printf("Error: Kernel too big\n");
        return 1;
    }

    int *hi = (int*)malloc(n * sizeof(int));
    int *ho = (int*)malloc(n * sizeof(int));
    int *hk = (int*)malloc(k * sizeof(int));
    
    printf("Enter %d input values:\n", n);
    for (int i = 0; i < n; i++) scanf("%d", &hi[i]);
    
    printf("Enter %d kernel values:\n", k);
    for (int i = 0; i < k; i++) scanf("%d", &hk[i]);
    
    int *di, *dout;
    hipMalloc(&di, n * sizeof(int));
    hipMalloc(&dout, n * sizeof(int));
    
    hipMemcpyToSymbol(HIP_SYMBOL(dk), hk, k * sizeof(int));
    hipMemcpy(di, hi, n * sizeof(int), hipMemcpyHostToDevice);
    
    int blks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    conv1d<<<blks, THREADS_PER_BLOCK>>>(di, dout, n, k);
    
    hipMemcpy(ho, dout, n * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("\nResult:\n");
    for (int i = 0; i < n; i++) {
        printf("%d ", ho[i]);
        if ((i+1) % 10 == 0) printf("\n");
    }
    printf("\n");
    
    free(hi); free(ho); free(hk);
    hipFree(di); hipFree(dout);
    
    return 0;
}