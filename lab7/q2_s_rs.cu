#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_STRING_LENGTH 1024

__global__ void constructRS(char *S, char *RS, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int startPos = idx * len - (idx * (idx - 1)) / 2;

    for (int i = 0; i < len - idx; i++) {
        RS[startPos + i] = S[i];
    }
}

int main(void) {
    char S[MAX_STRING_LENGTH];
    char *d_S, *d_RS;
    int len;

    printf("Enter a string: ");
    scanf("%s", S);
    len = strlen(S);

    int rsLen = (len * (len + 1)) / 2;

    char *RS = (char *)malloc(sizeof(char) * (rsLen + 1));
    RS[rsLen] = '\0';

    hipMalloc((void**)&d_S, sizeof(char) * (len + 1));
    hipMalloc((void**)&d_RS, sizeof(char) * (rsLen + 1));

    hipMemcpy(d_S, S, sizeof(char) * (len + 1), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    int blockSize = 256;
    int gridSize = (len + blockSize - 1) / blockSize;
    constructRS<<<gridSize, blockSize>>>(d_S, d_RS, len);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
    }

    hipMemcpy(RS, d_RS, sizeof(char) * (rsLen + 1), hipMemcpyDeviceToHost);

    printf("Input S: %s\n", S);
    printf("Output RS: %s\n", RS);
    printf("Time taken: %f milliseconds\n", elapsedTime);

    hipFree(d_S);
    hipFree(d_RS);
    free(RS);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}