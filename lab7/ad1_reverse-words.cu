#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_SENTENCE_LENGTH 1024

__global__ void reverseWords(char *sentence, int *wordStarts, int *wordEnds, int wordCount) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < wordCount) {
        int start = wordStarts[idx];
        int end = wordEnds[idx];

        while (start < end) {
            char temp = sentence[start];
            sentence[start] = sentence[end];
            sentence[end] = temp;
            start++;
            end--;
        }
    }
}

int main(void) {
    char sentence[MAX_SENTENCE_LENGTH];
    char *d_sentence;
    int *d_wordStarts, *d_wordEnds;
    int wordCount = 0;

    printf("Enter a sentence: ");
    fgets(sentence, MAX_SENTENCE_LENGTH, stdin);

    sentence[strcspn(sentence, "\n")] = '\0';

    int sentenceLength = strlen(sentence);

    int wordStarts[MAX_SENTENCE_LENGTH];
    int wordEnds[MAX_SENTENCE_LENGTH];

    
    for (int i = 0; i < sentenceLength; i++) {
        if (sentence[i] != ' ') {
            wordStarts[wordCount] = i;
            while (i < sentenceLength && sentence[i] != ' ') {
                i++;
            }
            wordEnds[wordCount] = i - 1;
            wordCount++;
        }
    }

    hipMalloc((void**)&d_sentence, sizeof(char) * (sentenceLength + 1));
    hipMalloc((void**)&d_wordStarts, sizeof(int) * wordCount);
    hipMalloc((void**)&d_wordEnds, sizeof(int) * wordCount);

    hipMemcpy(d_sentence, sentence, sizeof(char) * (sentenceLength + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_wordStarts, wordStarts, sizeof(int) * wordCount, hipMemcpyHostToDevice);
    hipMemcpy(d_wordEnds, wordEnds, sizeof(int) * wordCount, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (wordCount + blockSize - 1) / blockSize;
    reverseWords<<<gridSize, blockSize>>>(d_sentence, d_wordStarts, d_wordEnds, wordCount);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
    }

    hipMemcpy(sentence, d_sentence, sizeof(char) * (sentenceLength + 1), hipMemcpyDeviceToHost);

    printf("Reversed sentence: %s\n", sentence);

    hipFree(d_sentence);
    hipFree(d_wordStarts);
    hipFree(d_wordEnds);

    return 0;
}