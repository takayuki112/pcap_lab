#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_S 1024
#define MAX_W 64

__global__ void countWord(char *s, char *w, int sLen, int wLen, unsigned int *cnt) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i <= sLen - wLen) {
        bool match = true;
        for (int j = 0; j < wLen; j++) {
            if (s[i + j] != w[j]) {
                match = false;
                break;
            }
        }
        if (match) atomicAdd(cnt, 1);
    }
}

int main(void) {
    char s[MAX_S];
    char w[MAX_W];
    char *d_s, *d_w;
    unsigned int cnt = 0;
    unsigned int *d_cnt;

    printf("Enter a sentence: ");
    fgets(s, MAX_S, stdin);
    printf("Enter the word to count: ");
    scanf("%s", w);

    s[strcspn(s, "\n")] = '\0';

    int sLen = strlen(s);
    int wLen = strlen(w);

    hipMalloc((void**)&d_s, sizeof(char) * (sLen + 1));
    hipMalloc((void**)&d_w, sizeof(char) * (wLen + 1));
    hipMalloc((void**)&d_cnt, sizeof(unsigned int));

    hipMemcpy(d_s, s, sizeof(char) * (sLen + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, sizeof(char) * (wLen + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_cnt, &cnt, sizeof(unsigned int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    int bSize = 256;
    int gSize = (sLen + bSize - 1) / bSize;
    countWord<<<gSize, bSize>>>(d_s, d_w, sLen, wLen, d_cnt);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(&cnt, d_cnt, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("The word '%s' occurs %u times in the sentence.\n", w, cnt);
    printf("Time taken: %f milliseconds\n", time);

    hipFree(d_s);
    hipFree(d_w);
    hipFree(d_cnt);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}