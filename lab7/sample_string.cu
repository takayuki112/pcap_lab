#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "hip/hip_runtime.h"


#define N 1024

__global__ void cudacount(char *A, unsigned int *d_count) {
    int i = threadIdx.x;
    if (A[i] == 'a') atomicAdd(d_count, 1);
}

int main(void) {
    char A[N];
    char *dA;

    unsigned int count = 0;
    unsigned int *d_count, *result = (unsigned int*)malloc(sizeof(unsigned int));

    printf("Enter a string: ");
    scanf("%s", A);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipMalloc((void**)&dA, sizeof(char) * (strlen(A) + 1)); 
    hipMalloc((void**)&d_count, sizeof(unsigned int));

    hipMemcpy(dA, A, (strlen(A) + 1) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
        printf("CUDA Error1: %s\n", hipGetErrorString(error));

    cudacount<<<1, N>>>(dA, d_count);
    error = hipGetLastError();
    if (error != hipSuccess)
        printf("CUDA Error2: %s\n", hipGetErrorString(error));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("Total occurrences of 'a' = %u\n", *result); 
    printf("Time taken = %f\n", elapsedTime);

    hipFree(dA);
    hipFree(d_count);
    free(result);

    return 0;
}