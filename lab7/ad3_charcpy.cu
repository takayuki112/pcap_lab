#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_STRING_LENGTH 1024

__global__ void copyCharacter(char *input, char *output, int inputLength, int *outputOffsets) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < inputLength) {
        char currentChar = input[idx];
        int startPos = outputOffsets[idx];

        for (int i = 0; i < idx + 1; i++) {
            output[startPos + i] = currentChar;
        }
    }
}

int main(void) {
    char input[MAX_STRING_LENGTH];
    char *d_input, *d_output;
    int *d_outputOffsets;

    printf("Enter a string: ");
    scanf("%s", input);

    int inputLength = strlen(input);

    int outputLength = 0;
    for (int i = 0; i < inputLength; i++) {
        outputLength += (i + 1);
    }

    char *output = (char *)malloc(sizeof(char) * (outputLength + 1)); 
    output[outputLength] = '\0'; 

    int *outputOffsets = (int *)malloc(sizeof(int) * inputLength);
    outputOffsets[0] = 0;
    for (int i = 1; i < inputLength; i++) {
        outputOffsets[i] = outputOffsets[i - 1] + i;
    }

    hipMalloc((void**)&d_input, sizeof(char) * (inputLength + 1));
    hipMalloc((void**)&d_output, sizeof(char) * (outputLength + 1));
    hipMalloc((void**)&d_outputOffsets, sizeof(int) * inputLength);

    hipMemcpy(d_input, input, sizeof(char) * (inputLength + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_outputOffsets, outputOffsets, sizeof(int) * inputLength, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (inputLength + blockSize - 1) / blockSize;
    copyCharacter<<<gridSize, blockSize>>>(d_input, d_output, inputLength, d_outputOffsets);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
    }

    hipMemcpy(output, d_output, sizeof(char) * (outputLength + 1), hipMemcpyDeviceToHost);

    printf("Output: %s\n", output);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_outputOffsets);

    free(output);
    free(outputOffsets);

    return 0;
}