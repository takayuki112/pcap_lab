#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_STRING_LENGTH 1024

__global__ void concatenateString(char *input, char *output, int inputLength, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n) {
        int startPos = idx * inputLength;
        for (int i = 0; i < inputLength; i++) {
            output[startPos + i] = input[i];
        }
    }
}

int main(void) {
    char input[MAX_STRING_LENGTH];
    char *d_input, *d_output;
    int n;

    printf("Enter a string: ");
    scanf("%s", input);
    printf("Enter the number of times to concatenate: ");
    scanf("%d", &n);

    int inputLength = strlen(input);
    int outputLength = inputLength * n;

    char *output = (char *)malloc(sizeof(char) * (outputLength + 1)); 
    output[outputLength] = '\0'; 

    hipMalloc((void**)&d_input, sizeof(char) * (inputLength + 1));
    hipMalloc((void**)&d_output, sizeof(char) * (outputLength + 1));

    hipMemcpy(d_input, input, sizeof(char) * (inputLength + 1), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    concatenateString<<<gridSize, blockSize>>>(d_input, d_output, inputLength, n);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
    }

    hipMemcpy(output, d_output, sizeof(char) * (outputLength + 1), hipMemcpyDeviceToHost);

    printf("Output: %s\n", output);

    hipFree(d_input);
    hipFree(d_output);

    free(output);

    return 0;
}