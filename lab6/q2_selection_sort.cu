#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void sort_step(int *A, int *R, int N){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<N){
        // count the number of elements smaller than A[i], and put it in the correct place in R[i]
        int smaller = 0;
        for(int j= 0; j<N; j++){
            if(A[j] < A[i] || (A[j] == A[i] && j<i)) //extra logic for duplicate case
                smaller++;
        }
        R[smaller] = A[i];
    }
}

int main(void){
    int N;
    int *A, *R, *dA, *dR;

    printf("Enter size of the input array to be sorted: ");
    scanf("%d", &N);

    int size = N*sizeof(int);
    A = (int*)malloc(size);
    R = (int*)malloc(size);

    printf("Enter %d elements to sort: \n", N);
    for(int i=0; i<N; i++) scanf("%d", &A[i]);

    hipMalloc((void**)&dA, size);
    hipMalloc((void**)&dR, size);
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 8;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    sort_step<<<blocksPerGrid, threadsPerBlock>>>(dA, dR, N);

    hipMemcpy(R, dR, size, hipMemcpyDeviceToHost);

    printf("Sorted array:\n");
    for(int i=0; i<N; i++) printf("%d  ", R[i]);
    printf("\n");

    hipFree(dA);
    hipFree(dR);
    free(A);
    free(R);
    
}