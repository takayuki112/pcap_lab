#include <stdio.h>
#include <hip/hip_runtime.h>

#include <stdbool.h>

__global__ void swap_step(int *A, int N, bool even){ 
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<N-1 && (i%2==0) == even){
        if(A[i] > A[i+1]){
            int t = A[i];
            A[i] = A[i+1];
            A[i+1] = t;
        }
    }
}

int main(void){
    int N;
    int *A, *dA;

    printf("Enter size of the input array to be sorted: ");
    scanf("%d", &N);

    int size = N*sizeof(int);
    A = (int*)malloc(size);

    printf("Enter %d elements to sort: \n", N);
    for(int i=0; i<N; i++) scanf("%d", &A[i]);

    hipMalloc((void**)&dA, size);
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 8;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    //even-odd sorting logic
    for(int i=0; i<=N/2; i++){
        //check_swap_all_evens
        swap_step<<<threadsPerBlock, blocksPerGrid>>>(dA, N, true);
        //check_swap_all_odds
        swap_step<<<threadsPerBlock, blocksPerGrid>>>(dA, N, false);
    }
    hipMemcpy(A, dA, size, hipMemcpyDeviceToHost);

    printf("Sorted array:\n");
    for(int i=0; i<N; i++) printf("%d  ", A[i]);
    printf("\n");

    hipFree(dA);
    free(A);
    
}