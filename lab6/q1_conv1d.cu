#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void conv_1d(float *N, float *M, float *P, int Mask_width, int Width){
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    float Pvalue = 0;
    int N_start_point = i- (Mask_width/2);
    for(int j = 0; j < Mask_width; j++){
        if(N_start_point + j >= 0 && N_start_point + j < Width)
            Pvalue += N[N_start_point + j] * M[j];
    }
    P[i] = Pvalue;
}

int main(void){
    int Width = 7;
    int Mask_width = 5;

    printf("Enter Width and Mask-Width:\n");
    scanf("%d%d", &Width, &Mask_width);

    int sizeN = Width*sizeof(float);
    int sizeM = Mask_width*sizeof(float);

    float *N = (float*)malloc(sizeN);
    float *M = (float*)malloc(sizeM);
    float *P = (float*)malloc(sizeN);

    //Initialize values in M and N
    printf("Enter the %d elements in N\n", Width);
    for(int i=0; i<Width; i++) scanf("%f", &N[i]);

    printf("Enter the %d elements in Mask-M\n", Mask_width);
    for(int i=0; i<Mask_width; i++) scanf("%f", &M[i]);
    
    float *dN, *dM, *dP;
    hipMalloc((void**)&dN, sizeN);
    hipMalloc((void**)&dM, sizeM);
    hipMalloc((void**)&dP, sizeN);

    hipMemcpy(dN, N, sizeN, hipMemcpyHostToDevice);
    hipMemcpy(dM, M, sizeM, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (Width + threadsPerBlock - 1) / threadsPerBlock;
    
    conv_1d<<<blocksPerGrid, threadsPerBlock>>>(dN, dM, dP, Mask_width, Width);

    hipMemcpy(P, dP, sizeN, hipMemcpyDeviceToHost);

    printf("\nResult - P : \n");
    for(int i = 0; i < Width; i++)
        printf("%.1f \t", P[i]);

    printf("\n");

    hipFree(dN);
    hipFree(dM);
    hipFree(dP);
    free(N);
    free(M);
    free(P);
}