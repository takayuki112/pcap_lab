#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



__global__ void matmul_cols(int *a, int *b, int *c, int ha, int wahb, int wb){
    int colid = threadIdx.x;

    for(int rowid=0; rowid<ha; rowid++){
        int sum = 0;
        for(int k=0; k<wahb; k++) sum += a[rowid*wahb + k] * b[k*wb + colid];

        c[rowid*wb + colid] = sum;
    }
}


int main(void){
    int ha, wahb, wb, i;
    int *a, *b, *c, *da, *db, *dc;

    printf("Enter 3 dimensions ha, wa=hb, wb: \n");
    scanf("%d%d%d", &ha, &wahb, &wb);

    a = (int*)malloc(sizeof(int) * ha*wahb);
    b = (int*)malloc(sizeof(int) * wahb*wb);
    c = (int*)malloc(sizeof(int) * ha*wb);

    printf("Enter Matrix A (%d elements): \n", ha*wahb);
    for(i=0; i<ha*wahb; i++) scanf("%d", &a[i]);

    printf("Enter Matrix B (%d elements): \n", wahb*wb);
    for(i=0; i<wahb*wb; i++) scanf("%d", &b[i]);

    hipMalloc((void**)&da, sizeof(int)*ha*wahb);
    hipMalloc((void**)&db, sizeof(int)*wahb*wb);
    hipMalloc((void**)&dc, sizeof(int)*ha*wb);

    hipMemcpy(da, a, sizeof(int)*ha*wahb, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int)*wahb*wb, hipMemcpyHostToDevice);

    int blocksPerGrid = 1;
    int threadsPerBlock = wb;
    
    matmul_cols<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, ha, wahb, wb);

    hipMemcpy(c, dc, sizeof(int)*ha*wb, hipMemcpyDeviceToHost);
    
    printf("Resultant matrix: \n");
    for(i=0; i<ha*wb; i++){
        if(i%wb==0) printf("\n");
        printf("%d \t", c[i]);
    }

    hipFree(da); hipFree(db); hipFree(dc);
    free(a); free(b); free(c);


}
