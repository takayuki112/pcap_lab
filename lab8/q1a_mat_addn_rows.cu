#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void add_row(int *a, int *b, int *r, int m, int n){

    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(row_idx < m){
        for(int col = 0; col<n; col++) 
            r[n * row_idx + col] = a[n * row_idx + col] + b[n * row_idx + col];            
    }
}

int main(void){
    int *a, *b, *c, *da, *db, *dc;
    int m, n, i, j; // m rows, n columns

    printf("Enter the dimensions of the matrices: ");
    scanf("%d%d", &m, &n);

    a = (int*) malloc(sizeof(int) * m*n);
    b = (int*) malloc(sizeof(int) * m*n);
    c = (int*) malloc(sizeof(int) * m*n);

    printf("\nEnter %d x %d elements of matrix a: \n", m, n);
    for(i=0; i<m*n; i++) scanf("%d", &a[i]);

    printf("\nEnter %d x %d elements of matrix b: \n", m, n);
    for(i=0; i<m*n; i++) scanf("%d", &b[i]);

    hipMalloc((void**)&da, sizeof(int)*m*n);
    hipMalloc((void**)&db, sizeof(int)*m*n);
    hipMalloc((void**)&dc, sizeof(int)*m*n);

    hipMemcpy(da, a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int)*m*n, hipMemcpyHostToDevice);

    int threadsPerBlock = 16;
    int blocksPerGrid = (m + threadsPerBlock - 1)/threadsPerBlock;

    add_row<<<threadsPerBlock, blocksPerGrid>>>(da, db, dc, m, n);

    hipMemcpy(c, dc, sizeof(int)*m*n, hipMemcpyDeviceToHost);

    printf("\nResultant Matrix: \n");
    for(i=0; i<m; i++){
        for(j=0; j<n; j++) printf("%d\t", c[i*n + j]);
        printf("\n");
    }

}