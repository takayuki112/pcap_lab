#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void add_ele(int *a, int *b, int *r, int m, int n){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < m*n) r[idx] = a[idx] + b[idx];            
}

int main(void){
    int *a, *b, *c, *da, *db, *dc;
    int m, n, i, j; // m rows, n columns

    printf("Enter the dimensions of the matrices: ");
    scanf("%d%d", &m, &n);

    a = (int*) malloc(sizeof(int) * m*n);
    b = (int*) malloc(sizeof(int) * m*n);
    c = (int*) malloc(sizeof(int) * m*n);

    printf("\nEnter %d x %d elements of matrix a: \n", m, n);
    for(i=0; i<m*n; i++) scanf("%d", &a[i]);

    printf("\nEnter %d x %d elements of matrix b: \n", m, n);
    for(i=0; i<m*n; i++) scanf("%d", &b[i]);

    hipMalloc((void**)&da, sizeof(int)*m*n);
    hipMalloc((void**)&db, sizeof(int)*m*n);
    hipMalloc((void**)&dc, sizeof(int)*m*n);

    hipMemcpy(da, a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int)*m*n, hipMemcpyHostToDevice);

    int threadsPerBlock = 16;
    int blocksPerGrid = (m*n + threadsPerBlock - 1)/threadsPerBlock;

    add_ele<<<threadsPerBlock, blocksPerGrid>>>(da, db, dc, m, n);

    hipMemcpy(c, dc, sizeof(int)*m*n, hipMemcpyDeviceToHost);

    printf("\nResultant Matrix: \n");
    for(i=0; i<m; i++){
        for(j=0; j<n; j++) printf("%d\t", c[i*n + j]);
        printf("\n");
    }

}