#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void add_col(int *a, int *b, int *r, int m, int n){

    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(col_idx < n){
        for(int row = 0; row<n; row++) 
            r[n * row + col_idx] = a[n * row + col_idx] + b[n * row + col_idx];            
    }
}

int main(void){
    int *a, *b, *c, *da, *db, *dc;
    int m, n, i, j; // m rows, n columns

    printf("Enter the dimensions of the matrices: ");
    scanf("%d%d", &m, &n);

    a = (int*) malloc(sizeof(int) * m*n);
    b = (int*) malloc(sizeof(int) * m*n);
    c = (int*) malloc(sizeof(int) * m*n);

    printf("\nEnter %d x %d elements of matrix a: \n", m, n);
    for(i=0; i<m*n; i++) scanf("%d", &a[i]);

    printf("\nEnter %d x %d elements of matrix b: \n", m, n);
    for(i=0; i<m*n; i++) scanf("%d", &b[i]);

    hipMalloc((void**)&da, sizeof(int)*m*n);
    hipMalloc((void**)&db, sizeof(int)*m*n);
    hipMalloc((void**)&dc, sizeof(int)*m*n);

    hipMemcpy(da, a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int)*m*n, hipMemcpyHostToDevice);

    int threadsPerBlock = 16;
    int blocksPerGrid = (n + threadsPerBlock - 1)/threadsPerBlock;

    add_col<<<threadsPerBlock, blocksPerGrid>>>(da, db, dc, m, n);

    hipMemcpy(c, dc, sizeof(int)*m*n, hipMemcpyDeviceToHost);

    printf("\nResultant Matrix: \n");
    for(i=0; i<m; i++){
        for(j=0; j<n; j++) printf("%d\t", c[i*n + j]);
        printf("\n");
    }

}