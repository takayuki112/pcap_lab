#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void calc_rowsums(int *a, int *b, int m, int n){

    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(row_idx < m){
        int sum = 0;
        for(int col = 0; col<n; col++) sum+= a[n * row_idx + col];

        //replace all even numbered elements with rowsum
        for(int col = 0; col<n; col++){
            int ele_no = n*row_idx + col;
            if(ele_no %2 != 0) b[ele_no] = sum;
        }
    }
}

__global__ void calc_colsums(int *a, int *b, int m, int n){

    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(col_idx < n){
        int sum = 0;
        for(int row = 0; row<m; row++) sum+= a[n * row + col_idx];

        //replace all odd numbered elements with colsum !
        for(int row = 0; row<m; row++){
            int ele_no = n*row + col_idx;
            if(ele_no %2 == 0) b[ele_no] = sum;
        }
    }
}

int main(void){
    int *a, *b, *da, *db;
    int m, n, i, j; // m rows, n columns

    printf("Enter the dimensions of the matrices: ");
    scanf("%d%d", &m, &n);

    a = (int*) malloc(sizeof(int) * m*n);
    b = (int*) malloc(sizeof(int) * m*n);

    printf("\nEnter %d x %d elements of matrix a: \n", m, n);
    for(i=0; i<m*n; i++) scanf("%d", &a[i]);

    hipMalloc((void**)&da, sizeof(int)*m*n);
    hipMalloc((void**)&db, sizeof(int)*m*n);

    hipMemcpy(da, a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(db, da, sizeof(int)*m*n, hipMemcpyDeviceToDevice);

    int threadsPerBlock = 16;

    int blocksPerGrid = (m + threadsPerBlock - 1)/threadsPerBlock;
    calc_rowsums<<<threadsPerBlock, blocksPerGrid>>>(da, db, m, n);

    blocksPerGrid = (n + threadsPerBlock - 1)/threadsPerBlock;
    calc_colsums<<<threadsPerBlock, blocksPerGrid>>>(da, db, m, n);

    hipMemcpy(b, db, sizeof(int)*m*n, hipMemcpyDeviceToHost);

    printf("\nResultant Matrix B: \n");
    for(i=0; i<m; i++){
        for(j=0; j<n; j++) printf("%d\t", b[i*n + j]);
        printf("\n");
    }

}