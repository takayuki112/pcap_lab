#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ int fact(int n){
    if(n<=1) return 1;
    return n*fact(n-1);
}

__device__ int sum_digits(int n){
    int sum = 0;
    while(n!=0){
        sum += n%10;
        n/=10;
    }
    return sum;
}

__global__ void raplace(int *a, int *b, int n){
    int r = threadIdx.x;
    int c = threadIdx.y;

    if(r==c) b[n*r + c] = 0;                        //principal diagonal = 0
    if(r>c) b[n*r + c] = sum_digits(b[n*r + c]);    //below diagonal
    if(r<c) b[n*r + c] = fact(b[n*r + c]);          //above diag

}

int main(void){
    int *a, *b, *da, *db;
    int n, i, j; // m rows, n columns

    printf("Enter the dimension of the sqr matrix: ");
    scanf("%d", &n);

    a = (int*) malloc(sizeof(int) * n*n);
    b = (int*) malloc(sizeof(int) * n*n);

    printf("\nEnter %d x %d elements of matrix a: \n", n, n);
    for(i=0; i<n*n; i++) scanf("%d", &a[i]);

    hipMalloc((void**)&da, sizeof(int)*n*n);
    hipMalloc((void**)&db, sizeof(int)*n*n);

    hipMemcpy(da, a, sizeof(int)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(db, da, sizeof(int)*n*n, hipMemcpyDeviceToDevice);

    
    int blocksPerGrid = 1;
    dim3 threadsPerBlock(n, n, 1);

    raplace<<<blocksPerGrid, threadsPerBlock>>>(da, db, n);

    hipMemcpy(b, db, sizeof(int)*n*n, hipMemcpyDeviceToHost);

    printf("\nResultant Matrix B: \n");
    for(i=0; i<n; i++){
        for(j=0; j<n; j++) printf("%d\t", b[i*n + j]);
        printf("\n");
    }

}