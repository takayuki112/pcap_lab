#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void add(int *a, int *b, int *c, int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x; // Calculate index
    printf("Thread idx started= %d \n", index);
    if(index < N) {
        c[index] = a[index] + b[index];  // Element-wise addition
    }
    printf("Thread idx finished = %d \n", index);
}

int main(void){
    int N = 64;  
    int size = N * sizeof(int);
    
    int *a, *b, *c;  
    int *da, *db, *dc;  

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    // Initialize (host) vectors
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 3;
    }

    // Allocate device memory
    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, size);
    hipMalloc((void**)&dc, size);

    // Copy data from host to device
    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size, hipMemcpyHostToDevice);

    // Set up kernel launch parameters
    int threadsPerBlock = 64;
    // int blocksPerGrid = N / 256;        // this doesn't work if N is not divisible by 256! - some get left out
    // int blocksPerGrid = (N+255) / 256;  //that'sw why we round up N with a +255 - to make sure we have enough blocks!
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the add kernel - a single grid
    add<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, N);

    // Copy the result back to host
    hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

    printf("First few Results in c...\n");
    printf(" a  \t + \t b  \t = \t c \n");
    printf("------------------------------------\n");
    for (int i = 0; i < 16; i++) {  
        printf(" %d \t + \t %d \t = \t %d\n", a[i], b[i], c[i]);
    }

    // Free memory
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);
    
    return 0;
}
