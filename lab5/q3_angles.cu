//write a simple cuda program to find the sine of N angles stored in an array
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"

__global__ void calcSine(float *angles, float *sineResults, int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        sineResults[index] = sinf(angles[index]);  
    }
}

int main(void) {
    int N = 1024;  
    int size = N * sizeof(float);

    float *angles, *sineResults;     
    float *d_angles, *d_sineResults; 

    // Allocate host memory
    angles = (float*)malloc(size);
    sineResults = (float*)malloc(size);

    // Initialize some angles (in radians)
    for (int i = 0; i < N; i++) {
        angles[i] = i * 0.01;  
    }

    // Allocate memory
    hipMalloc((void**)&d_angles, size);
    hipMalloc((void**)&d_sineResults, size);

    hipMemcpy(d_angles, angles, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    calcSine<<<blocksPerGrid, threadsPerBlock>>>(d_angles, d_sineResults, N);

    hipMemcpy(sineResults, d_sineResults, size, hipMemcpyDeviceToHost);

    printf("First few Results...\n");
    printf("Index \t Angle (radians) \t Sine of angle\n");
    printf("---------------------------------------------\n");
    for (int i = 0; i < 16; i++) {  
        printf("%d \t %f \t\t %f\n", i, angles[i], sineResults[i]);
    }

    // Free memory
    hipFree(d_angles);
    hipFree(d_sineResults);
    free(angles);
    free(sineResults);

    return 0;
}
