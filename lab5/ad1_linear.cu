#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void applyLinearFunction(float *x, float *y, float w, float b, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        y[index] = w * x[index] + b;  
    }
}

int main(void) {
    int N = 1024;  
    int size = N * sizeof(float);

    float *x, *y;      
    float *d_x, *d_y;  

    // Define w and b
    float w = 2.0f;  
    float b = 1.0f;  

    x = (float*)malloc(size);
    y = (float*)malloc(size);

    // Initialize input array
    for (int i = 0; i < N; i++) {
        x[i] = i * 0.01;  
    }

    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    applyLinearFunction<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, w, b, N);

    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    printf("First few results...\n");
    printf("Index \t x \t\t y = wx + b\n");
    printf("---------------------------------------------\n");
    for (int i = 0; i < 16; i++) {  
        printf("%d \t %f \t %f\n", i, x[i], y[i]);
    }

    // Free memory
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    return 0;
}
