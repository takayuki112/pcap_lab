#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"

__global__ void swapMinAheadOfMe(int* a, int N){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (i <= N) {
        int minidx = i;
        for (int j = i + 1; j < N; j++) {
            if (a[j] < a[minidx]) {
                minidx = j;
            }
        }
        
        if (minidx != i) {
            int temp = a[i];
            a[i] = a[minidx];
            a[minidx] = temp;
        }
        
        printf("Thread %d: Swapped %d with %d, coz i = %d and minidx = %d\n", i, a[i], a[minidx], i, minidx);
    }
}

int main() {
    int N = 10;
    int h_a[N] = {29, 10, 14, 37, 13, 6, 23, 12, 44, 19};
    for (int i = 0; i < N; i++) {
        printf("%d ", h_a[i]);
    }
    printf("\n");

    int* d_a;
    hipMalloc(&d_a, N * sizeof(int));
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = N;
    int gridSize = 1;
    swapMinAheadOfMe<<<gridSize, blockSize>>>(d_a, N);

    hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_a[i]);
    }
    printf("\n");

    hipFree(d_a);

    return 0;
}
