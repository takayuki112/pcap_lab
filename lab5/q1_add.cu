#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void add(int *a, int *b, int *c, int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x; // Calculate index
    if(index < N) {
        c[index] = a[index] + b[index];  // Element-wise addition
    }
}

int main(void){
    int N = 2048;  
    int size = N * sizeof(int);
    
    int *a, *b, *c;  
    int *da, *db, *dc;  

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    // Initialize (host) vectors
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, size);
    hipMalloc((void**)&dc, size);

    // Copy data from host to device
    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size, hipMemcpyHostToDevice);

    // Set up kernel launch parameters
    // a.
    // int threadsPerBlock = N;
    // int blocksPerGrid = 1; 

    //b.
    int threadsPerBlock = 1;
    int blocksPerGrid = N;    //Fun fact - if you put a number less than N, then the remaining values in C will remain 0

    // Launch the add kernel - a single grid
    add<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, N);

    // Copy the result back to host
    hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

    printf("First few Results in c...\n");
    printf(" a  \t + \t b  \t = \t c \n");
    printf("------------------------------------\n");
    for (int i = 0; i < 16; i++) {  
        printf(" %d \t + \t %d \t = \t %d\n", a[i], b[i], c[i]);
    }

    // Free memory
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);
    
    return 0;
}
