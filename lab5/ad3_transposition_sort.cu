#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void oddEvenSortStep(float *arr, int N, int phase) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int i = index * 2 + phase;  

    if (i < N - 1 && arr[i] > arr[i + 1]) {
        float temp = arr[i];
        arr[i] = arr[i + 1];
        arr[i + 1] = temp;
    }
}

void oddEvenSort(float *arr, int N) {
    float *d_arr;
    size_t size = N * sizeof(float);
    
    hipMalloc((void **)&d_arr, size);
    hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N / 2 + threadsPerBlock - 1) / threadsPerBlock;

    for (int step = 0; step < N; step++) {
        oddEvenSortStep<<<blocksPerGrid, threadsPerBlock>>>(d_arr, N, step % 2);
        hipDeviceSynchronize();
    }

    hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main() {
    int N = 16;
    float arr[N];

    printf("\nUnsorted array:\n");
    for (int i = 0; i < N; i++) {
        arr[i] = (float)(rand() % 100);
        printf("%.1f ", arr[i]);
    }
    printf("\n");

    oddEvenSort(arr, N);

    printf("\nSorted array:\n");
    for (int i = 0; i < N; i++) {
        printf("%.1f ", arr[i]);
    }
    printf("\n");

    return 0;
}
