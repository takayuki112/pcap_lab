// write a cuda program to perform sparse matrix-vector multiplication, using csr storage format. 
// represent the input matrix as csr in the host code.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void matrix_csr(int *A, int m, int n, int** data, int** col_idx, int** row_ptr, int* n_non0) {
    int nz_count = 0;
    for(int i=0; i<m*n; i++) if(A[i]!=0) nz_count++;
    
    *data = (int*)malloc(sizeof(int)*nz_count);
    *col_idx = (int*)malloc(sizeof(int)*nz_count);
    *row_ptr = (int*)malloc(sizeof(int)*(m+1)); 
    
    (*row_ptr)[0] = 0;
    nz_count = 0;
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            int ele = A[n*i + j];
            if(ele != 0) {
                (*data)[nz_count] = ele;
                (*col_idx)[nz_count] = j;
                nz_count++;
            }
        }
        (*row_ptr)[i+1] = nz_count;
    }
    
    *n_non0 = nz_count;
}

__global__ void spmv_csr_kernel(int m, int *row_ptr, int *col_idx, int *data, int *x, int *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m) {
        int sum = 0;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row + 1];
        
        for (int i = row_start; i < row_end; i++) {
            sum += data[i] * x[col_idx[i]];
        }
        
        y[row] = sum;
    }
}

int main(void){
    int m, n, i, j;
    printf("Enter dimensions m and n: ");
    scanf("%d%d", &m, &n);

    int *A = (int*)malloc(m*n*sizeof(int));
    int *v = (int*)malloc(n*sizeof(int));

    printf("Enter the %d x %d entries of the matrix A:\n", m, n);
    for(i=0; i<m; i++){
        for(j=0; j<n; j++) scanf("%d", &A[n*i + j]);
    }

    printf("Enter the %d dimensional vector:\n", n);
    for(i=0; i<n; i++) scanf("%d", &v[i]);

    int *data, *col_idx, *row_ptr, n_non0;
    matrix_csr(A, m, n, &data, &col_idx, &row_ptr, &n_non0);

    int *d_data, *d_col_idx, *d_row_ptr, *d_v, *d_result;
    hipMalloc((void**)&d_data, n_non0 * sizeof(int));
    hipMalloc((void**)&d_col_idx, n_non0 * sizeof(int));
    hipMalloc((void**)&d_row_ptr, (m+1) * sizeof(int));
    hipMalloc((void**)&d_v, n * sizeof(int));
    hipMalloc((void**)&d_result, m * sizeof(int));
    
    hipMemcpy(d_data, data, n_non0 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, col_idx, n_non0 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, row_ptr, (m+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, n * sizeof(int), hipMemcpyHostToDevice);
    
    int *result = (int*)malloc(m * sizeof(int));

    int blockSize = 256;
    int numBlocks = (m + blockSize - 1) / blockSize;
    spmv_csr_kernel<<<numBlocks, blockSize>>>(m, d_row_ptr, d_col_idx, d_data, d_v, d_result);
    
    hipMemcpy(result, d_result, m * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Result vector:\n");
    for (i = 0; i < m; i++) {
        printf("%d ", result[i]);
    }
    printf("\n");
    
    free(A); free(v); free(data); free(col_idx); free(row_ptr); free(result);
    hipFree(d_data); hipFree(d_col_idx); hipFree(d_row_ptr); hipFree(d_v); hipFree(d_result);
    
    return 0; 

}