#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ int comp(int n) {
    int b = 0;
    int t = n;
    while (t > 0) {
        b++;
        t >>= 1;
    }
    
    if (b == 0) return 0;
    
    int m = (1 << b) - 1;
    return n ^ m;
}

__global__ void repNB(int *m, int r, int c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= r * c) return;
    
    int row = i / c;
    int col = i % c;
    
    if (row == 0 || row == r - 1 || col == 0 || col == c - 1) {
        return;
    }
    
    m[i] = comp(m[i]);
}

int main(void) {
    int *m, *d;
    int r, c;
    
    printf("Enter the number of rows and columns: ");
    scanf("%d %d", &r, &c);
    
    m = (int*)malloc(r * c * sizeof(int));
    
    printf("Enter the matrix elements (%d x %d):\n", r, c);
    for (int i = 0; i < r * c; i++) {
        scanf("%d", &m[i]);
    }
    
    hipMalloc((void**)&d, r * c * sizeof(int));
    hipMemcpy(d, m, r * c * sizeof(int), hipMemcpyHostToDevice);
    
    int tpb = 16;
    int bpg = (r * c + tpb - 1) / tpb;
    
    repNB<<<bpg, tpb>>>(d, r, c);
    
    hipMemcpy(m, d, r * c * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("\nResultant Matrix:\n");
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            int v = m[i * c + j];
            
            if (i == 0 || i == r - 1 || j == 0 || j == c - 1) {
                printf("%d\t", v);
            } else {
                int t = v;
                int b = 0;
                while (t > 0) {
                    b++;
                    t >>= 1;
                }
                b = (b == 0) ? 1 : b;
                
                for (int k = b - 1; k >= 0; k--) {
                    printf("%d", (v >> k) & 1);
                }
                printf("\t");
            }
        }
        printf("\n");
    }
    
    hipFree(d);
    free(m);
    
    return 0;
}