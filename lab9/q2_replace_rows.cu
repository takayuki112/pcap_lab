#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

__device__ int power(int a, int b){
    int r = 1;
    for(int i=0; i<b; i++) r*=a;
    return r;
}

__global__ void rep_row(int *a, int m, int n){

    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(row_idx < m){
        for(int col = 0; col<n; col++) 
            a[n * row_idx + col] = power(a[n * row_idx + col], row_idx+1) ;            
    }
}

int main(void){
    int *a, *da;
    int m, n, i, j; // m rows, n columns

    printf("Enter the dimensions of the matrices: ");
    scanf("%d%d", &m, &n);

    a = (int*) malloc(sizeof(int) * m*n);

    printf("\nEnter %d x %d elements of matrix a: \n", m, n);
    for(i=0; i<m*n; i++) scanf("%d", &a[i]);


    hipMalloc((void**)&da, sizeof(int)*m*n);
    hipMemcpy(da, a, sizeof(int)*m*n, hipMemcpyHostToDevice);

    int threadsPerBlock = 16;
    int blocksPerGrid = (m + threadsPerBlock - 1)/threadsPerBlock;

    rep_row<<<threadsPerBlock, blocksPerGrid>>>(da, m, n);

    hipMemcpy(a, da, sizeof(int)*m*n, hipMemcpyDeviceToHost);
    printf("\nResultant Matrix: \n");
    for(i=0; i<m; i++){
        for(j=0; j<n; j++) printf("%d\t", a[i*n + j]);
        printf("\n");
    }

}