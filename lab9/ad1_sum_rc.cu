#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void calcRowColSum(int *in, int *out, int r, int c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= r * c) return;
    
    int row = idx / c;
    int col = idx % c;
    
    int sum = 0;
    
    for (int j = 0; j < c; j++) sum += in[row * c + j];
    for (int i = 0; i < r; i++) sum += in[i * c + col];
    
    out[idx] = sum;
}

int main(void) {
    int *a, *r, *da, *dr;
    int m, n;
    
    printf("Enter the number of rows and columns: ");
    scanf("%d %d", &m, &n);
    
    a = (int*)malloc(m * n * sizeof(int));
    r = (int*)malloc(m * n * sizeof(int));
    
    printf("Enter the matrix elements (%d x %d):\n", m, n);
    for (int i = 0; i < m * n; i++) {
        scanf("%d", &a[i]);
    }
    
    hipMalloc((void**)&da, m * n * sizeof(int));
    hipMalloc((void**)&dr, m * n * sizeof(int));
    
    hipMemcpy(da, a, m * n * sizeof(int), hipMemcpyHostToDevice);
    
    int tpb = 256;
    int bpg = (m * n + tpb - 1) / tpb;
    
    calcRowColSum<<<bpg, tpb>>>(da, dr, m, n);
    
    hipMemcpy(r, dr, m * n * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("\nResultant Matrix (sum of row and column for each element):\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d\t", r[i * n + j]);
        }
        printf("\n");
    }
    
    hipFree(da);
    hipFree(dr);
    free(a);
    free(r);
    
    return 0;
}